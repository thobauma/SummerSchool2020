#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "util.hpp"

// naive solution:
__global__
void reverse_stringNaiveSol(char* str, int n){
    int i = threadIdx.x;
    if (i<n){
        char tmp = str[i];
        str[i] = str[n-i-1];
        str[n-i-1] = tmp;
    }
}
// reverse_stringNaiveSol<<<1,1024>>>(string,n);


// shared version:
__global__
void reverse_stringSharedSol(char* str, int n){
    __shared__ char tmp[1024];
    int i = threadIdx.x;
    if (i<n){
        char tmp[i] = str[i];
        __syncthreads();
        str[i] = tmp[n-i-1];
    }
}

// optimal solution
__global__
void reverse_stringOptimalSol(char* str, int n){
    int i = threadIdx.x;
    if (i<n/2){
        char tmp = str[i];
        str[i] = str[n-i-1];
        str[n-i-1] = tmp;
    }
}


// TODO : implement a kernel that reverses a string of length n in place
// MY VERSION
__global__
void reverse_string(char* str, int n){
    extern __shared__ char stringBuffer[];
    auto i = threadIdx.x;
    // auto i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<n){
        stringBuffer[i]=str[i];
    }
    __syncthreads();
    str[n-i-1] = stringBuffer[i];
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // TODO : call the string reverse function
    auto block_dim = n;
    auto grid_dim = (n + block_dim - 1) / block_dim;
    reverse_stringOptimalSol<<<1,1024>>>(string,n);
    // reverse_string<<<1, block_dim, (n + 1) * sizeof(char)>>>(string, n);
    // reverse_string<<<grid_dim, block_dim, (n + 1) * sizeof(char)>>>(string, n);
    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

